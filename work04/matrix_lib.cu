#include "hip/hip_runtime.h"
#include "matrix_lib.h"

int global_n_block = 4096;
int global_n_thread = 256;
hipError_t hipError_t;

__global__ 
void scalar_aux(int fim, float *rows, float scalar_value)
{
  int i;
  int ini = blockIdx.x * blockDim.x + threadIdx.x;
  int passo = blockDim.x * gridDim.x;

  for (i = ini; i < fim; i += passo) 
  {
    rows[i] = rows[i] * scalar_value;
  }
}

__global__
void matrix_matrix_mult_aux(int widthA, int widthB, int heightA, int heightB, float *d_rowsA, float *d_rowsB, float *d_rowsC)
{
  int i, j, index, step;

  index = blockIdx.x * blockDim.x + threadIdx.x;
  step = gridDim.x * gridDim.x;

  for (i = index; i < heightA * widthB; i += step)
  {
    d_rowsC[i] = 0;

    for (j = 0; j < widthA; j++)
    {
      d_rowsC[i] += d_rowsA[widthA * (i / heightA) + j] * d_rowsB[(i % heightA) + j * widthB];
    }
  }
}

int scalar_matrix_mult(float scalar_value, struct matrix *matrix)
{
  if(matrix == NULL)
  {
    return ERRO;
  }

  int size = matrix->height * matrix->width;
  int i;

  int end = (size + DATASET_SIZE - 1) / DATASET_SIZE;
  int tam = DATASET_SIZE;

  for(i = 0; i < end; ++i)
  {
    if(size % DATASET_SIZE != 0 && i == end - 1)
    {
      tam = size % DATASET_SIZE;
    }
    
    hipError_t = hipMemcpy(matrix->d_rows, matrix->h_rows+(i*tam), tam*sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess) 
    {
      printf("hipMemcpy (h -> d) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
      return ERRO;
    }

    int blockSize = global_n_thread;
    int numBlocks = (tam + blockSize - 1) / blockSize;
    
    if (numBlocks > global_n_block)
    {
      numBlocks = global_n_block;
    }

    scalar_aux<<<numBlocks, blockSize>>>(tam, matrix->d_rows, scalar_value);

    hipDeviceSynchronize();

    hipError_t = hipMemcpy(matrix->h_rows+(i*tam), matrix->d_rows, tam*sizeof(float), hipMemcpyDeviceToHost);
  
    if (hipError_t != hipSuccess)
    {
      printf("hipMemcpy (d -> h) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
      
      return ERRO;
    }
  }

  return SUCESSO;
}

int matrix_matrix_mult(struct matrix *matrixA, struct matrix *matrixB, struct matrix *matrixC)
{
  int numBlocks;
  int blockSize;
  int matrixC_size;

  if (matrixA == NULL ||
      matrixB == NULL ||
      matrixC == NULL)
  {
    return ERRO;
  }

  if (matrixA->d_rows == NULL ||
      matrixB->d_rows == NULL ||
      matrixC->d_rows == NULL)
  {
    return ERRO;
  }

  if (matrixA->width != matrixB->height)
  {
    return ERRO;
  }

  if (matrixA->height != matrixC->height ||
      matrixB->width != matrixC->width)
  {
    return ERRO;
  }

  matrixC_size = matrixC->height * matrixC->width;

  blockSize = global_n_thread;
  numBlocks = (matrixC_size + blockSize - 1) / blockSize;
    
  if (numBlocks > global_n_block)
  {
    numBlocks = global_n_block;
  }

  matrix_matrix_mult_aux<<<numBlocks, blockSize>>>(matrixA->width, matrixB->width, matrixA->height, matrixB->height, matrixA->d_rows, matrixB->d_rows, matrixC->d_rows);
  
  hipDeviceSynchronize();

  return SUCESSO;
}

int set_grid_size(int threads_per_block, int max_blocks_per_grid)
{
  if(threads_per_block > MAX_THREAD ||
     max_blocks_per_grid > MAX_BLOCK) 
  {
    return ERRO;
  }

  global_n_block = threads_per_block;
  global_n_thread = max_blocks_per_grid;
  
  return SUCESSO;
}