#include "hip/hip_runtime.h"
#include "matrix_lib.h"

int global_n_block = 4096;
int global_n_thread = 256;
hipError_t hipError_t;

__global__ 
void scalar_aux(int fim, float *rows, float scalar_value)
{
  int i;
  int ini = blockIdx.x * blockDim.x + threadIdx.x;
  int passo = blockDim.x * gridDim.x;

  for (i = ini; i < fim; i += passo) 
  {
    rows[i] = rows[i] * scalar_value;
  }
}

int scalar_matrix_mult(float scalar_value, struct matrix *matrix)
{
  if(matrix == NULL)
  {
    return ERRO;
  }

  int size = matrix->height * matrix->width;
  int i;

  int end = (size + DATASET_SIZE - 1) / DATASET_SIZE;
  int tam = DATASET_SIZE;

  for(i = 0; i < end; ++i)
  {
    if(size % DATASET_SIZE != 0 && i == end - 1)
    {
      tam = size % DATASET_SIZE;
    }
    
    hipError_t = hipMemcpy(matrix->d_rows, matrix->h_rows+(i*tam), tam*sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess) 
    {
      printf("hipMemcpy (h -> d) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
      return ERRO;
    }

    int blockSize = global_n_thread;
    int numBlocks = (tam + blockSize - 1) / blockSize;
    
    if (numBlocks > global_n_block)
    {
      numBlocks = global_n_block;
    }

    scalar_aux<<<numBlocks, blockSize>>>(tam, matrix->d_rows, scalar_value);

    hipDeviceSynchronize();

    hipError_t = hipMemcpy(matrix->h_rows+(i*tam), matrix->d_rows, tam*sizeof(float), hipMemcpyDeviceToHost);
  
    if (hipError_t != hipSuccess)
    {
      printf("hipMemcpy (d -> h) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
      
      return ERRO;
    }
  }

  return SUCESSO;
}

int matrix_matrix_mult(struct matrix *matrixA, struct matrix *matrixB, struct matrix *matrixC)
{
    return SUCESSO;
}

int set_grid_size(int threads_per_block, int max_blocks_per_grid)
{
  if(threads_per_block > MAX_THREAD ||
     max_blocks_per_grid > MAX_BLOCK) 
  {
    return ERRO;
  }

  global_n_block = threads_per_block;
  global_n_thread = max_blocks_per_grid;
  
  return SUCESSO;
}